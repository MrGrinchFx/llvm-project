
#include <hip/hip_runtime.h>
// Tests CUDA compilation with -S and -emit-llvm.

// RUN: %clang -### -S --target=x86_64-linux-gnu --cuda-gpu-arch=sm_20 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck -check-prefix HOST -check-prefix SM20 %s
// RUN: %clang -### -S --target=x86_64-linux-gnu --cuda-host-only -o foo.s --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck -check-prefix HOST %s
// RUN: %clang -### -S --target=x86_64-linux-gnu --cuda-gpu-arch=sm_20 \
// RUN:   --cuda-device-only -o foo.s --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck -check-prefix SM20 %s
// RUN: %clang -### -S --target=x86_64-linux-gnu --cuda-gpu-arch=sm_20 \
// RUN:   --cuda-gpu-arch=sm_30 --cuda-device-only --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck -check-prefix SM20 -check-prefix SM30 %s

// HOST-DAG: "-cc1" "-triple" "x86_64-unknown-linux-gnu"
// SM20-DAG: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// SM20-same: "-target-cpu" "sm_20"
// SM30-DAG: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// SM30-same: "-target-cpu" "sm_30"

// RUN: not %clang -### -S --target=x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-gpu-arch=sm_20 --cuda-gpu-arch=sm_30 -o foo.s %s 2>&1 \
// RUN:   | FileCheck -check-prefix MULTIPLE-OUTPUT-FILES %s
// MULTIPLE-OUTPUT-FILES: error: cannot specify -o when generating multiple output files
// Make sure we do not get duplicate diagnostics.
// MULTIPLE-OUTPUT-FILES-NOT: error: cannot specify -o when generating multiple output files
